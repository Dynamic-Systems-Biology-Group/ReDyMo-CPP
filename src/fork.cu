#include "hip/hip_runtime.h"
#include "fork.cuh"
#include <stdio.h>

typedef struct _transcription_region_t
{
    int start;
    int end;
} transcription_region_t;

__device__ unsigned int int_rand(unsigned int *state)
{
    printf("line: Random %d\n", __LINE__);
    unsigned int x = *state;
    // 32-bit XOR Shift RNG
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;

    *state = x;
    return x;
}

__device__ double uniform_rand(unsigned int *state)
{
    printf("line: uniform rand %d\n", __LINE__);
    return ((double)int_rand(state)) / UINT_MAX;
}

__device__ bool collided(int transcription_period,
                         int transcription_regions_size,
                         const transcription_region_t *transcription_regions, int at,
                         int direction, int time)
{
    printf("line: check collision %d\n", __LINE__);
    for (int i = 0; i < transcription_regions_size; i++)
    {
        printf("line: for each transc. reg. %d\n", __LINE__);
        transcription_region_t region = transcription_regions[i];

        int t_dir   = 1;
        int t_start = region.start;
        int t_end   = region.end;

        int replisome_position_within_region = at - t_start;

        if (region.end < region.start)
        {
            printf("line: resersed region %d\n", __LINE__);
            t_dir   = -1;
            t_start = region.end;
            t_end   = region.start;
        }

        // If inside transcription region
        if (!(t_start <= at && at <= t_end)) continue;

        printf("line: inside transc region %d\n", __LINE__);
        replisome_position_within_region = t_end - at;
        int RNAP_position =
            t_dir == 1 ? time % transcription_period
                       : transcription_period - (time % transcription_period);
        int replisome_position =
            t_dir == 1
                ? replisome_position_within_region % transcription_period
                : transcription_period -
                      (replisome_position_within_region % transcription_period);

        return replisome_position == RNAP_position && direction != t_dir;
    }
    return false;
}
__device__ int get_global_id()
{
    return blockIdx.x * blockDim.x + threadIdx.x;
}
/**
 *
 */
__global__ void fork(uint seed)
{
    uint seed_boggled;
    seed_boggled = seed ^ get_global_id();
    printf("[GPU] Hi there from %d, here is a rand %u\n", get_global_id(), int_rand(&seed_boggled));
}
