#include "hip/hip_runtime.h"
#include "fork.cuh"
#include <stdio.h>

__device__ unsigned int int_rand(unsigned int *state)
{
    unsigned int x = *state;
    // 32-bit XOR Shift RNG
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;

    *state = x;
    return x;
}

__device__ double uniform_rand(unsigned int *state)
{
    return ((double)int_rand(state)) / UINT_MAX;
}

__device__ bool collided(int transcription_period,
                         int transcription_regions_size,
                         const transcription_region_t *transcription_regions,
                         int at, int direction, int time)
{
    for (int i = 0; i < transcription_regions_size; i++)
    {
        transcription_region_t region = transcription_regions[i];

        int t_dir   = 1;
        int t_start = region.start;
        int t_end   = region.end;

        int replisome_position_within_region = at - t_start;

        if (region.end < region.start)
        {
            t_dir   = -1;
            t_start = region.end;
            t_end   = region.start;
        }

        // If inside transcription region
        if (!(t_start <= at && at <= t_end)) continue;

        replisome_position_within_region = t_end - at;
        int RNAP_position =
            t_dir == 1 ? time % transcription_period
                       : transcription_period - (time % transcription_period);
        int replisome_position =
            t_dir == 1
                ? replisome_position_within_region % transcription_period
                : transcription_period -
                      (replisome_position_within_region % transcription_period);

        return replisome_position == RNAP_position && direction != t_dir;
    }
    return false;
}
__device__ int get_global_id() { return blockIdx.x * blockDim.x + threadIdx.x; }

__global__ void cuda_fork(
    // End Time
    int *end_time,
    // Number of already replicated bases
    int *replicated,
    // Number of free forks
    int *free_forks,
    // Start Location and Direction Arrays
    int *start_locations, int *start_directions,
    // Replication-transcription collision counts
    int *rt_collisions,
    // Replication timestamps for base pairs
    unsigned int *replication_times,
    // Repliction initiation probability landscape
    const float *probability_landscape,
    // Genome chromosome boundaries
    const int *chromosome_boundaries,
    // Transcription period
    const int transcription_period,
    // Transcription region count
    const int transcription_regions_size,
    // Transcription regions
    const transcription_region_t *transcription_regions,
    // Timeout
    const unsigned int timeout,
    // Total genome base pairs
    const int genome_size,
    // Chromosome count
    const int chromosome_count,
    // Number of max forks
    const int max_forks,
    // Initial seed for random
    const unsigned long seed,
    // Colldown after detach before next attempt
    const int fork_cooldown)
{
    // Replication fork ID, make manager thread -1
    int fork_id = get_global_id() - 1;

    // Local time
    int time = 0;

    // Fork cooldown
    int cooldown = 0;

    // Fork location
    int at = -1;

    // Fork direction
    int direction = 0;

    // Next chromosome boundary
    int boundary = -1;

    // Replicated
    int local_replicated = 0;

    // If fork is free
    bool free = true;

    // Initial RNG state
    unsigned int state = ((unsigned int)fork_id + 1) * seed;

    // Do until entire genome is replicated
    while (time < timeout && (*replicated) < genome_size)
    {
        __syncthreads();

        time++;

        // Try to attach forks to genome (fork manager)
        if (fork_id < 0 && (*free_forks) > 1)
        {
            const int prev_free = *free_forks;
            int free_cnt        = prev_free;

            for (int attempt = 0; attempt < prev_free && free_cnt > 1;
                 attempt++)
            {
                unsigned int location = int_rand(&state) % (genome_size - 1);

                // Check if replicated and check probability
                if (!replication_times[location] &&
                    uniform_rand(&state) < probability_landscape[location])
                {
                    printf("Attaching forks\n");
                    int i = 0;

                    // Spawn forks

                    // Fork to -1
                    for (bool attached = false; i < max_forks && !attached; i++)
                    {
                        if (start_locations[i] == -1)
                        {
                            // TODO: the actual base replicated is the next one
                            start_locations[i]  = location + 1;
                            start_directions[i] = -1;

                            attached = true;

                            break;
                        }
                    }

                    // Fork to +1
                    for (bool attached = false; i < max_forks && !attached; i++)
                    {
                        if (start_locations[i] == -1)
                        {
                            start_locations[i]  = location;
                            start_directions[i] = 1;

                            attached = true;

                            break;
                        }
                    }

                    atomicSub(free_forks, 2);
                    free_cnt -= 2;
                }
            }
        }

        // If not attached, check for start_location
        if (!cooldown && start_locations[fork_id] > -1 && at < 0)
        {
            at        = start_locations[fork_id];
            free      = false;
            direction = start_directions[fork_id];

            // Find relevant chromosome boundary
            for (int c = 0; c < chromosome_count + 1; c++)
            {
                if (chromosome_boundaries[c] > at)
                {
                    if (direction < 0)
                        boundary = chromosome_boundaries[c - 1];
                    else
                        boundary = chromosome_boundaries[c];
                }
            }
        }

        // If attached, loop copy until machinery release
        if (!cooldown && at >= 0)
        {
            printf("%d replicating at %d\n", fork_id, at);
            int next = at + direction;

            bool on_boundary = at < 0 ? next == boundary : at == boundary;

            bool collision =
                collided(transcription_period, transcription_regions_size,
                         transcription_regions, at, direction, time);

            if (collision) atomicAdd(rt_collisions, 1);
            // If has not reached a boundary, collided with transcription
            // machinery or hit an already replicated base, continue
            if (next != boundary + direction && !collision &&
                !atomicCAS(&replication_times[next], 0, time))
            {
                local_replicated++;
                at = next;
            }
            // Set cooldown and detach fork
            else
            {
                atomicAdd(replicated, local_replicated);
                local_replicated = 0;
                cooldown         = fork_cooldown;
                at               = -1;
                float pct_rep    = (float)(*replicated) / genome_size;
                printf("Replicated: %d / %d %f%\n", *replicated, genome_size,
                       pct_rep * 100);
            }
        }

        // Fork Grace period before attaching again
        if (at < 0)
        {
            if (cooldown)
                cooldown--;
            else if (!free)
            {
                atomicAdd(free_forks, 1);
                start_locations[fork_id] = -1;
                free                     = true;
            }
        }

        __syncthreads();
    }

    if (!free) atomicAdd(free_forks, 1);

    // TODO: maybe use the largest time with cmpexch
    (*end_time) = time;
    if (fork_id < 0 && time >= timeout) printf("Timeout\n");
}
