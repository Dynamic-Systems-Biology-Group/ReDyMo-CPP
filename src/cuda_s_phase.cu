#include "cuda_s_phase.hpp"
#include "fork.cuh"
#include "util.hpp"
#include <iostream>

#ifdef CUDA_ENABLED

CUDASPhase::CUDASPhase(Configuration &configuration,
                       std::shared_ptr<DataProvider> data,
                       unsigned long long seed)
    : data(data)
{
    auto args = configuration.arguments();

    origins_range        = args.constitutive;
    n_resources          = args.resources;
    replication_speed    = args.speed;
    timeout              = args.timeout;
    transcription_period = args.period;
    has_dormant          = args.dormant;
    organism             = args.organism;
    name                 = args.name;
    output_folder        = args.output;

    checkpoint_times.start_create = std::chrono::steady_clock::now();

    std::vector<std::shared_ptr<Chromosome>> chromosomes;

    auto codes = data->get_codes();
    for (auto code = codes.begin(); code != codes.end(); code++)
    {
        chromosomes.push_back(std::make_shared<Chromosome>(*code, data));
    }

    genome = std::make_shared<Genome>(chromosomes, seed);

    fork_manager =
        std::make_shared<ForkManager>(n_resources, genome, replication_speed);

    checkpoint_times.end_create = std::chrono::steady_clock::now();
}

CUDASPhase::~CUDASPhase() {}

void CUDASPhase::simulate(int sim_number)
{

    checkpoint_times.start_sim = std::chrono::steady_clock::now();

    int alpha                     = 1;
    int time                      = 0;
    int constitutive_origins      = (int)genome->n_constitutive_origins();
    int n_collisions              = 0;
    bool use_constitutive_origins = origins_range > 0;

    std::cout << "[INFO] Preparing CUDA simulation " << sim_number << std::endl;

    std::cout << "[INFO] Loading data to GPU" << std::endl;

    //////////////////////////
    // Single value buffers //
    //////////////////////////
    int *d_end_time, *d_replicated, *d_free_forks;
    hipMalloc(&d_end_time, sizeof(int));
    hipMalloc(&d_replicated, sizeof(int));
    hipMalloc(&d_free_forks, sizeof(int));

    hipMemset(d_end_time, 0, sizeof(int));
    hipMemset(d_replicated, 0, sizeof(int));
    hipMemcpy(d_free_forks, &n_resources, sizeof(int), hipMemcpyHostToDevice);

    ////////////////////////////////
    // Start locations/directions //
    ////////////////////////////////
    int *d_start_locations, *d_start_directions;
    hipMalloc(&d_start_locations, n_resources * sizeof(int));
    hipMalloc(&d_start_directions, n_resources * sizeof(int));

    hipMemset(d_start_locations, -1, n_resources * sizeof(int));
    hipMemset(d_start_directions, 0, n_resources * sizeof(int));

    /////////////////////////////
    // Collision count buffers //
    /////////////////////////////
    int *d_rt_collisions;
    hipMalloc(&d_rt_collisions, genome->size() * sizeof(int));

    hipMemset(d_rt_collisions, 0, genome->size() * sizeof(int));

    ///////////////////////////////////////////
    // Replication timestamps for base pairs //
    ///////////////////////////////////////////
    unsigned int *d_replication_times;
    hipMalloc(&d_replication_times, genome->size() * sizeof(unsigned int));

    hipMemset(d_replication_times, 0, genome->size() * sizeof(unsigned int));

    ///////////////////////////////////////////
    // Probability landscape for full genome //
    ///////////////////////////////////////////
    float *d_probability_landscape;
    hipMalloc(&d_probability_landscape, genome->size() * sizeof(float));

    std::vector<float> probability_landscape;
    for (auto chrm : genome->chromosomes)
    {
        std::vector<double> chrm_probabilities =
            data->get_probability_landscape(chrm->get_code());
        probability_landscape.insert(probability_landscape.end(),
                                     chrm_probabilities.begin(),
                                     chrm_probabilities.end());
    }
    hipMemcpy(d_probability_landscape, probability_landscape.data(),
               probability_landscape.size() * sizeof(float),
               hipMemcpyHostToDevice);

    /////////////////////////////////////////////////////////
    // Boundaries of chromosomes (for all the flat arrays) //
    /////////////////////////////////////////////////////////
    int *d_chromosome_boundaries;
    int boundary_count = genome->chromosomes.size() + 1;

    hipMalloc(&d_chromosome_boundaries, boundary_count * sizeof(int));

    std::vector<int> boundaries(boundary_count, 0);

    // Generate boundaries list
    for (long int c = 0, bd = -1; c < boundary_count; c++)
    {
        boundaries[c] = bd;
        // TODO: if a bug appears, check this logic
        if (c < genome->chromosomes.size())
            bd += genome->chromosomes[c]->size();
    }
    hipMemcpy(d_chromosome_boundaries, boundaries.data(),
               boundary_count * sizeof(int), hipMemcpyHostToDevice);

    ///////////////////////////
    // Transcription regions //
    ///////////////////////////
    transcription_region_t *d_transcription_regions;
    std::vector<transcription_region_t> transcription_regions_v;

    for (int c = 0; c < genome->chromosomes.size(); c++)
    {
        auto regions = genome->chromosomes[c]->get_transcription_regions();

        for (int r = 0; r < regions->size(); r++)
        {
            transcription_regions_v.push_back((*regions)[r]);
        }
    }

    hipMalloc(&d_transcription_regions,
               transcription_regions_v.size() * sizeof(transcription_region_t));

    hipMemcpy(d_transcription_regions, transcription_regions_v.data(),
               transcription_regions_v.size() * sizeof(transcription_region_t),
               hipMemcpyHostToDevice);

    int *d_workers_running;
    hipMalloc(&d_workers_running, sizeof(int));

    hipMemset(d_workers_running, 0, sizeof(int));

    ///////////////////
    // Create kernel //
    ///////////////////
    hipDeviceSynchronize();
    std::cout << "[INFO] Launching GPU simulation " << sim_number << std::endl;
    // Add an extra thread for management
    cuda_fork<<<1, n_resources + 1>>>(
        d_end_time, d_replicated, d_free_forks, d_workers_running,
        d_start_locations, d_start_directions, d_rt_collisions,
        d_replication_times, d_probability_landscape, d_chromosome_boundaries,
        transcription_period, transcription_regions_v.size(),
        d_transcription_regions, timeout, genome->size(),
        genome->chromosomes.size(), n_resources, genome->seed, 0);

    // Wait kernel end
    hipDeviceSynchronize();
    // TODO: add hipFree or use hipDeviceReset()
}

#endif
