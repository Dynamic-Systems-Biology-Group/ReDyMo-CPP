#include "fork.cuh"
#include "cuda_s_phase.hpp"
#include <iostream>

#ifdef CUDA_ENABLED

// CUDASPhase::CUDASPhase(int origins_range, int n_resources,
//                      int replication_speed, int timeout,
//                      int transcription_period, bool has_dormant,
//                      std::shared_ptr<DataProvider> data, std::string organism,
//                      std::string name, std::string output_folder,
//                      unsigned long long seed)
//     : origins_range(origins_range), n_resources(n_resources),
//       replication_speed(replication_speed), timeout(timeout),
//       transcription_period(transcription_period), has_dormant(has_dormant),
//       data(data), organism(organism), name(name), output_folder(output_folder),
// {
//     checkpoint_times.start_create = std::chrono::steady_clock::now();

//     std::vector<std::shared_ptr<Chromosome>> chromosomes;

//     auto codes = data->get_codes();
//     for (auto code = codes.begin(); code != codes.end(); code++)
//     {
//         chromosomes.push_back(std::make_shared<Chromosome>(*code, data));
//     }

//     genome = std::make_shared<Genome>(chromosomes, seed);

//     fork_manager =
//         std::make_shared<ForkManager>(n_resources, genome, replication_speed);

//     checkpoint_times.end_create = std::chrono::steady_clock::now();
// }

CUDASPhase::~CUDASPhase() {
}

void CUDASPhase::simulate(int sim_number){
    std::cout << "Will run CUDA simulation " << sim_number << std::endl;
    fork<<<1,100>>>(1234);
    hipDeviceSynchronize();
}

#endif
