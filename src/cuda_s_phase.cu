#include "cuda_s_phase.hpp"
#include "fork.cuh"
#include "util.hpp"
#include <iostream>

#ifdef CUDA_ENABLED

CUDASPhase::CUDASPhase(Configuration &configuration,
                       std::shared_ptr<DataProvider> data,
                       unsigned long long seed)
    : data(data)
{
    auto args = configuration.arguments();

    origins_range        = args.constitutive;
    n_resources          = args.resources;
    replication_speed    = args.speed;
    timeout              = args.timeout;
    transcription_period = args.period;
    has_dormant          = args.dormant;
    organism             = args.organism;
    name                 = args.name;
    output_folder        = args.output;

    checkpoint_times.start_create = std::chrono::steady_clock::now();

    std::vector<std::shared_ptr<Chromosome>> chromosomes;

    auto codes = data->get_codes();
    for (auto code = codes.begin(); code != codes.end(); code++)
    {
        chromosomes.push_back(std::make_shared<Chromosome>(*code, data));
    }

    genome = std::make_shared<Genome>(chromosomes, seed);

    fork_manager =
        std::make_shared<ForkManager>(n_resources, genome, replication_speed);

    checkpoint_times.end_create = std::chrono::steady_clock::now();
}

CUDASPhase::~CUDASPhase() {}

void CUDASPhase::simulate(int sim_number)
{
    std::cout << "Will run CUDA simulation " << sim_number << std::endl;
    fork<<<1, 100>>>(1234);
    hipDeviceSynchronize();

    checkpoint_times.start_sim = std::chrono::steady_clock::now();

    int alpha                     = 1;
    int time                      = 0;
    int constitutive_origins      = (int)genome->n_constitutive_origins();
    int n_collisions              = 0;
    bool use_constitutive_origins = origins_range > 0;

    std::cout << "[INFO] Starting CUDA simulation " << sim_number << std::endl;

    std::cout << "[INFO] Loading data to GPU" << std::endl;

    //////////////////////////
    // Single value buffers //
    //////////////////////////
    int *end_time, *replicated, *free_forks;
    hipMalloc(&end_time, sizeof(int));
    hipMalloc(&replicated, sizeof(int));
    hipMalloc(&free_forks, sizeof(int));

    hipMemset(end_time, 0, sizeof(int));
    hipMemset(replicated, 0, sizeof(int));
    hipMemset(free_forks, n_resources, n_resources * sizeof(int));

    ////////////////////////////////
    // Start locations/directions //
    ////////////////////////////////
    int *start_locations, *start_directions;
    hipMalloc(&start_locations, n_resources * sizeof(int));
    hipMalloc(&start_directions, n_resources * sizeof(int));

    hipMemset(start_locations, -1, n_resources * sizeof(int));
    hipMemset(start_directions, 0, n_resources * sizeof(int));

    /////////////////////////////
    // Collision count buffers //
    /////////////////////////////
    int *rt_collisions;
    hipMalloc(&rt_collisions, genome->size() * sizeof(int));

    hipMemset(rt_collisions, 0, genome->size() * sizeof(int));

    ///////////////////////////////////////////
    // Replication timestamps for base pairs //
    ///////////////////////////////////////////
    int *replication_times;
    hipMalloc(&replication_times, genome->size() * sizeof(unsigned int));

    hipMemset(replication_times, 0, genome->size() * sizeof(int));

    ///////////////////////////////////////////
    // Probability landscape for full genome //
    ///////////////////////////////////////////
    float *probability_landscape;
    hipMalloc(&probability_landscape, genome->size() * sizeof(float));

    // TODO: make a flat probability landsacpe for entire genome
    // std::vector<float> probabilities = data->get_probability_landscape();
    // hipMemcpy(replication_times, probabilities.data(), sizeof(int), );

    hipMemset(probability_landscape, 0.f, genome->size() * sizeof(float));

    /////////////////////////////////////////////////////////
    // Boundaries of chromosomes (for all the flat arrays) //
    /////////////////////////////////////////////////////////
    int *chromosome_boundaries;
    hipMalloc(&chromosome_boundaries, genome->size() * sizeof(int));

    std::vector<int> boundaries(genome->chromosomes.size() + 1, 0);

    // Generate boundaries list
    for (long unsigned int c = 0, bd = -1; c < genome->chromosomes.size() + 1;
         c++)
    {
        boundaries[c] = bd;
        // TODO: if a bug appears, check this logic
        if (c < genome->chromosomes.size())
            bd += genome->chromosomes[c]->size();
    }
    hipMemcpy(replication_times, boundaries.data(),
               boundaries.size() * sizeof(int), hipMemcpyHostToDevice);

    ///////////////////////////
    // Transcription regions //
    ///////////////////////////
    transcription_region_t *transcription_regions;
    std::vector<transcription_region_t> transcription_regions_v;

    for (int c = 0; c < genome->chromosomes.size(); c++)
    {
        auto regions = genome->chromosomes[c]->get_transcription_regions();

        for (int r = 0; r < regions->size(); r++)
        {
            transcription_regions_v.push_back((*regions)[r]);
        }
    }

    hipMalloc(&transcription_regions,
               transcription_regions_v.size() * sizeof(transcription_region_t));

    hipMemcpy(replication_times, boundaries.data(),
               transcription_regions_v.size() * sizeof(transcription_region_t),
               hipMemcpyHostToDevice);

    ///////////////////
    // Create kernel //
    ///////////////////
    // Add an extra thread for management
    cuda_fork<<<1, n_resources + 1>>>();
    // TODO: add hipFree or use hipDeviceReset()
}

#endif
